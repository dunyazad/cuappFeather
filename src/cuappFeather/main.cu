#include "hip/hip_runtime.h"
#include "main.cuh"

__global__ void hello_kernel() {
	printf("Hello from CUDA kernel!\\n");
}

void TestCUDA()
{
	hello_kernel << <1, 1 >> > ();
	hipDeviceSynchronize();
	std::cout << "CUDA finished\\n";
}

//#include <Serialization.hpp>

#include <nvtx3/nvToolsExt.h>

//int cuMain(const std::vector<float3>& host_points);
#include <hip/hip_runtime.h>
#include <>
#include <nvtx3/nvToolsExt.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <iostream>
#include <unordered_map>
#include <unordered_set>
#include <vector>
#include <algorithm>

namespace Clustering
{
	struct Voxel
	{
		//float3 position;
		unsigned int label;
	};

	__global__ void Kernel_ClearVoxels(
		Voxel* d_voxels,
		unsigned int numberOfVoxels,
		dim3 volumeDimensions,
		float voxelSize,
		float3 volumeMin,
		float3 volumeCenter)
	{
		unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadid >= volumeDimensions.x * volumeDimensions.y * volumeDimensions.z) return;

		//d_voxels[threadid].position = make_float3(FLT_MAX, FLT_MAX, FLT_MAX);
		d_voxels[threadid].label = 0;
	}

	void ClearVoxels(
		Voxel* d_voxels,
		unsigned int numberOfVoxels,
		dim3 volumeDimensions,
		float voxelSize,
		float3 volumeMin,
		float3 volumeCenter)
	{
		nvtxRangePushA("ClearVoxels");

		unsigned int blockSize = 256;
		unsigned int gridSize = (numberOfVoxels + blockSize - 1) / blockSize;
		Kernel_ClearVoxels << <gridSize, blockSize >> > (d_voxels, numberOfVoxels, volumeDimensions, voxelSize, volumeMin, volumeCenter);

		hipDeviceSynchronize();
		nvtxRangePop();
	}

	__global__ void Kernel_OccupyVoxels(
		float* d_points,
		unsigned int numberOfPoints,
		Voxel* d_voxels,
		unsigned int numberOfVoxels,
		dim3 volumeDimensions,
		float voxelSize,
		float3 volumeMin,
		float3 volumeCenter,
		uint3* occupiedVoxelIndices,
		unsigned int* numberOfOccupiedVoxelIndices,
		unsigned int* occupiedPointIndices,
		unsigned int* numberOfOccupiedPointIndices)
	{
		unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadid >= numberOfPoints) return;

		auto gx = d_points[threadid * 3];
		auto gy = d_points[threadid * 3 + 1];
		auto gz = d_points[threadid * 3 + 2];

		if (gx < volumeMin.x || gx > volumeMin.x + volumeDimensions.x * voxelSize ||
			gy < volumeMin.y || gy > volumeMin.y + volumeDimensions.y * voxelSize ||
			gz < volumeMin.z || gz > volumeMin.z + volumeDimensions.z * voxelSize)
		{
			return;
		}

		unsigned int ix = (unsigned int)floorf((gx - volumeMin.x) / voxelSize);
		unsigned int iy = (unsigned int)floorf((gy - volumeMin.y) / voxelSize);
		unsigned int iz = (unsigned int)floorf((gz - volumeMin.z) / voxelSize);

		if (ix >= volumeDimensions.x || iy >= volumeDimensions.y || iz >= volumeDimensions.z) return;

		unsigned int volumeIndex = iz * volumeDimensions.x * volumeDimensions.y + iy * volumeDimensions.x + ix;
		auto& voxel = d_voxels[volumeIndex];

		//voxel.position.x = volumeMin.x + ix * voxelSize;
		//voxel.position.y = volumeMin.y + iy * voxelSize;
		//voxel.position.z = volumeMin.z + iz * voxelSize;
		voxel.label = volumeIndex;

		//alog("%f, %f, %f\n", voxel.position.x, voxel.position.y, voxel.position.z);

		auto voxelIndex = atomicAdd(numberOfOccupiedVoxelIndices, 1);
		occupiedVoxelIndices[voxelIndex] = make_uint3(ix, iy, iz);

		auto pointIndex = atomicAdd(numberOfOccupiedPointIndices, 1);
		occupiedPointIndices[pointIndex] = threadid;

		//alog("%d\n", index);
	}

	void OccupyVoxels(
		float* d_points,
		unsigned int numberOfPoints,
		Voxel* d_voxels,
		unsigned int numberOfVoxels,
		dim3 volumeDimensions,
		float voxelSize,
		float3 volumeMin,
		float3 volumeCenter,
		uint3* occupiedVoxelIndices,
		unsigned int* numberOfOccupiedVoxelIndices,
		unsigned int* occupiedPointIndices,
		unsigned int* numberOfOccupiedPointIndices)
	{
		nvtxRangePush("OccupyVoxels");

		unsigned int blockSize = 256;
		unsigned int gridSize = (numberOfPoints + blockSize - 1) / blockSize;

		Kernel_OccupyVoxels << <gridSize, blockSize >> > (
			d_points,
			numberOfPoints,
			d_voxels,
			numberOfVoxels,
			volumeDimensions,
			voxelSize,
			volumeMin,
			volumeCenter,
			occupiedVoxelIndices,
			numberOfOccupiedVoxelIndices,
			occupiedPointIndices,
			numberOfOccupiedPointIndices);

		hipDeviceSynchronize();
		nvtxRangePop();
	}

	__device__ __forceinline__ unsigned int FindRoot(Voxel* voxels, unsigned int idx) {
		while (true) {
			unsigned int parent = voxels[idx].label;
			unsigned int grand = voxels[parent].label;
			if (parent == idx) break;
			if (parent != grand) voxels[idx].label = grand;
			idx = parent;
		}
		return idx;
	}

	__device__ __forceinline__ void Union(Voxel* voxels, unsigned int a, unsigned int b) {
		unsigned int rootA = FindRoot(voxels, a);
		unsigned int rootB = FindRoot(voxels, b);
		if (rootA != rootB) {
			if (rootA < rootB)
				atomicMin(&voxels[rootB].label, rootA);
			else
				atomicMin(&voxels[rootA].label, rootB);
		}
	}

	__global__ void Kernel_InterBlockMerge(
		Voxel* voxels,
		uint3* occupiedIndices,
		unsigned int numOccupied,
		dim3 dims)
	{
		unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid >= numOccupied) return;

		uint3 idx = occupiedIndices[tid];
		unsigned int center = idx.z * dims.y * dims.x + idx.y * dims.x + idx.x;
		//if (voxels[center].position.x == FLT_MAX) return;
		if (0 == voxels[center].label) return;

		for (int dz = -1; dz <= 1; dz++) {
			int nz = idx.z + dz;
			if (nz < 0 || nz >= dims.z) continue;
			for (int dy = -1; dy <= 1; dy++) {
				int ny = idx.y + dy;
				if (ny < 0 || ny >= dims.y) continue;
				for (int dx = -1; dx <= 1; dx++) {
					int nx = idx.x + dx;
					if (nx < 0 || nx >= dims.x) continue;
					if (dx == 0 && dy == 0 && dz == 0) continue;

					unsigned int neighbor = nz * dims.y * dims.x + ny * dims.x + nx;
					//if (voxels[neighbor].position.x != FLT_MAX) {
					if (0 != voxels[neighbor].label) {
						Union(voxels, center, neighbor);
					}
				}
			}
		}
	}

	__global__ void Kernel_InterBlockMergeP(
		Voxel* voxels,
		uint3* occupiedIndices,
		unsigned int numOccupied,
		dim3 dims)
	{
		unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid >= numOccupied) return;

		uint3 idx = occupiedIndices[tid];
		unsigned int center = idx.z * dims.y * dims.x + idx.y * dims.x + idx.x;
		//if (voxels[center].position.x == FLT_MAX) return;
		if (0 == voxels[center].label) return;

		for (int dz = 0; dz <= 1; dz++) {
			int nz = idx.z + dz;
			if (nz < 0 || nz >= dims.z) continue;
			for (int dy = 0; dy <= 1; dy++) {
				int ny = idx.y + dy;
				if (ny < 0 || ny >= dims.y) continue;
				for (int dx = 0; dx <= 1; dx++) {
					int nx = idx.x + dx;
					if (nx < 0 || nx >= dims.x) continue;
					if (dx == 0 && dy == 0 && dz == 0) continue;

					unsigned int neighbor = nz * dims.y * dims.x + ny * dims.x + nx;
					//if (voxels[neighbor].position.x != FLT_MAX) {
					if (0 != voxels[neighbor].label) {
						Union(voxels, center, neighbor);
					}
				}
			}
		}
	}

	__global__ void Kernel_InterBlockMergeN(
		Voxel* voxels,
		uint3* occupiedIndices,
		unsigned int numOccupied,
		dim3 dims)
	{
		unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid >= numOccupied) return;

		uint3 idx = occupiedIndices[tid];
		unsigned int center = idx.z * dims.y * dims.x + idx.y * dims.x + idx.x;
		//if (voxels[center].position.x == FLT_MAX) return;
		if (0 == voxels[center].label) return;

		for (int dz = -1; dz <= 0; dz++) {
			int nz = idx.z + dz;
			if (nz < 0 || nz >= dims.z) continue;
			for (int dy = -1; dy <= 0; dy++) {
				int ny = idx.y + dy;
				if (ny < 0 || ny >= dims.y) continue;
				for (int dx = -1; dx <= 0; dx++) {
					int nx = idx.x + dx;
					if (nx < 0 || nx >= dims.x) continue;
					if (dx == 0 && dy == 0 && dz == 0) continue;

					unsigned int neighbor = nz * dims.y * dims.x + ny * dims.x + nx;
					//if (voxels[neighbor].position.x != FLT_MAX) {
					if (0 != voxels[neighbor].label) {
						Union(voxels, center, neighbor);
					}
				}
			}
		}
	}

	__global__ void Kernel_CompressLabels(Voxel* voxels, unsigned int N) {
		unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid >= N) return;

		//if (voxels[tid].position.x != FLT_MAX) {
		if (0 != voxels[tid].label) {
			voxels[tid].label = FindRoot(voxels, tid);
		}
	}

	void ConnectedComponentLabeling(
		Voxel* d_voxels,
		uint3* occupiedVoxelIndices,
		unsigned int numberOfOccupiedVoxelIndices,
		dim3 volumeDimensions)
	{
		unsigned int totalVoxels = volumeDimensions.x * volumeDimensions.y * volumeDimensions.z;
		unsigned int blockSize = 256;
		unsigned int gridVoxels = (totalVoxels + blockSize - 1) / blockSize;
		unsigned int gridOccupied = (numberOfOccupiedVoxelIndices + blockSize - 1) / blockSize;

		Kernel_InterBlockMerge << <gridOccupied, blockSize >> > (d_voxels, occupiedVoxelIndices, numberOfOccupiedVoxelIndices, volumeDimensions);
		hipDeviceSynchronize();

		//Kernel_InterBlockMergeP << <gridOccupied, blockSize >> > (d_voxels, occupiedVoxelIndices, numberOfOccupiedVoxelIndices, volumeDimensions);
		//hipDeviceSynchronize();

		//Kernel_InterBlockMergeN << <gridOccupied, blockSize >> > (d_voxels, occupiedVoxelIndices, numberOfOccupiedVoxelIndices, volumeDimensions);
		//hipDeviceSynchronize();

		//Kernel_CompressLabels << <gridVoxels, blockSize >> > (d_voxels, totalVoxels);
		//hipDeviceSynchronize();
	}

__global__ void Kernel_GetLabels(
	float* d_points,
	unsigned int numberOfPoints,
	Voxel* d_voxels,
	unsigned int numberOfVoxels,
	dim3 volumeDimensions,
	float voxelSize,
	float3 volumeMin,
	float3 volumeCenter,
	unsigned int* d_labels)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= numberOfPoints) return;

	auto gx = d_points[threadid * 3];
	auto gy = d_points[threadid * 3 + 1];
	auto gz = d_points[threadid * 3 + 2];

	if (gx < volumeMin.x || gx > volumeMin.x + volumeDimensions.x * voxelSize ||
		gy < volumeMin.y || gy > volumeMin.y + volumeDimensions.y * voxelSize ||
		gz < volumeMin.z || gz > volumeMin.z + volumeDimensions.z * voxelSize)
	{
		return;
	}

	unsigned int ix = (unsigned int)floorf((gx - volumeMin.x) / voxelSize);
	unsigned int iy = (unsigned int)floorf((gy - volumeMin.y) / voxelSize);
	unsigned int iz = (unsigned int)floorf((gz - volumeMin.z) / voxelSize);

	if (ix >= volumeDimensions.x || iy >= volumeDimensions.y || iz >= volumeDimensions.z) return;

	unsigned int volumeIndex = iz * volumeDimensions.x * volumeDimensions.y + iy * volumeDimensions.x + ix;
	auto& voxel = d_voxels[volumeIndex];

	d_labels[threadid] = voxel.label;
}

std::vector<unsigned int> GetLabels(
	float* d_points,
	unsigned int numberOfPoints,
	Voxel* d_voxels,
	unsigned int numberOfVoxels,
	dim3 volumeDimensions,
	float voxelSize,
	float3 volumeMin,
	float3 volumeCenter)
{
	unsigned int* d_labels = nullptr;
	hipMalloc(&d_labels, sizeof(unsigned int) * numberOfPoints);
	hipMemset(d_labels, -1, sizeof(unsigned int) * numberOfPoints);
	hipDeviceSynchronize();

	unsigned int blockSize = 256;
	unsigned int gridSize = (numberOfPoints + blockSize - 1) / blockSize;

	Kernel_GetLabels << <gridSize, blockSize >> > (
		d_points,
		numberOfPoints,
		d_voxels,
		numberOfVoxels,
		volumeDimensions,
		voxelSize,
		volumeMin,
		volumeCenter,
		d_labels);

	hipDeviceSynchronize();
	nvtxRangePop();

	std::vector<unsigned int> result(numberOfPoints);
	hipMemcpy(result.data(), d_labels, sizeof(unsigned int) * numberOfPoints, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	hipFree(d_labels);

	return result;
}

void VisualizeVoxels(
	Voxel* d_voxels,
	unsigned int numberOfVoxels,
	dim3 volumeDimensions,
	float voxelSize,
	float3 volumeMin)
{
	nvtxRangePush("VisualizeVoxels");

	Voxel* h_voxels = new Voxel[numberOfVoxels];
	hipMemcpy(h_voxels, d_voxels, sizeof(Voxel) * numberOfVoxels, hipMemcpyDeviceToHost);

	std::unordered_map<unsigned int, std::tuple<unsigned char, unsigned char, unsigned char>> labelToColor;

	std::unordered_map<unsigned int, unsigned int> labelHistogram;

	for (size_t i = 0; i < numberOfVoxels; i++)
	{
		auto& voxel = h_voxels[i];

		//if (voxel.position.x != FLT_MAX) // Only visualize occupied voxels
		if (0 != voxel.label) // Only visualize occupied voxels
		{
			unsigned int label = voxel.label;

			// Assign a unique color per label using a hash function
			if (labelToColor.find(label) == labelToColor.end())
			{
				unsigned char r = (label * 53) % 256;
				unsigned char g = (label * 97) % 256;
				unsigned char b = (label * 151) % 256;
				labelToColor[label] = std::make_tuple(r, g, b);
			}

			// Get the assigned color
			auto [r, g, b] = labelToColor[label];

			// Visualize the voxel with the computed color
/*               VD::AddCube("labeled voxels", { voxel.position.x, voxel.position.y, voxel.position.z },
				   0.05f, { r, g, b, 255 });*/

			if (0 == labelHistogram.count(voxel.label))
			{
				labelHistogram[voxel.label] = 1;
			}
			else
			{
				labelHistogram[voxel.label] += 1;
			}
		}
	}

	int i = 0;
	for (auto& [label, count] : labelHistogram)
	{
		alog("[%4d] voxel label - %16d : count - %8d\n", i++, label, count);
	}
	alog("\n");

	delete[] h_voxels;

	hipDeviceSynchronize();
	nvtxRangePop();
}

struct ClusteringCacheInfo
{
	float voxelSize;
	dim3 cacheDimensions;
	unsigned int numberOfVoxels;
	float3 cacheMin;

	hipArray* cacheData3D = nullptr;
	hipSurfaceObject_t surfaceObject3D;

	uint3* occupiedVoxelIndices;
	unsigned int* numberOfOccupiedVoxelIndices;
};
}


std::vector<unsigned int> cuMain(const std::vector<float3>& host_points, float3 center)
{
	nvtxRangePush("TestClustering");

	float* d_points = nullptr;
	hipMalloc(&d_points, sizeof(float) * host_points.size() * 3);
	hipMemcpy(d_points, host_points.data(), sizeof(float) * host_points.size() * 3, hipMemcpyHostToDevice);

	unsigned int numberOfPoints = host_points.size();
	//dim3 volumeDimensions(200, 300, 400);
	dim3 volumeDimensions(400, 400, 400);
	unsigned int numberOfVoxels = volumeDimensions.x * volumeDimensions.y * volumeDimensions.z;
	float voxelSize = 0.1f;
	//float3 volumeCenter = make_float3(3.9904f, -15.8357f, -7.2774f);
	//float3 volumeCenter = make_float3(4.0f, -15.0f, -7.0f);
	float3 volumeCenter = make_float3(center.x, center.y, center.z);
	float3 volumeMin = make_float3(
		volumeCenter.x - (float)(volumeDimensions.x / 2) * voxelSize,
		volumeCenter.y - (float)(volumeDimensions.y / 2) * voxelSize,
		volumeCenter.z - (float)(volumeDimensions.z / 2) * voxelSize);

	Clustering::Voxel* d_voxels = nullptr;
	hipMalloc(&d_voxels, sizeof(Clustering::Voxel) * numberOfVoxels);

	uint3* occupiedVoxelIndices = nullptr;
	hipMalloc(&occupiedVoxelIndices, sizeof(uint3) * 5000000);
	unsigned int* numberOfOccupiedVoxelIndices = nullptr;
	hipMalloc(&numberOfOccupiedVoxelIndices, sizeof(unsigned int));
	hipMemset(numberOfOccupiedVoxelIndices, 0, sizeof(unsigned int));

	unsigned int* occupiedPointIndices = nullptr;
	hipMalloc(&occupiedPointIndices, sizeof(unsigned int) * 5000000);
	unsigned int* numberOfOccupiedPointIndices = nullptr;
	hipMalloc(&numberOfOccupiedPointIndices, sizeof(unsigned int));
	hipMemset(numberOfOccupiedPointIndices, 0, sizeof(unsigned int));

	hipDeviceSynchronize();

	for (size_t i = 0; i < 10; i++)
	{
		hipMemset(numberOfOccupiedVoxelIndices, 0, sizeof(unsigned int));
		hipMemset(numberOfOccupiedPointIndices, 0, sizeof(unsigned int));

		ClearVoxels(d_voxels, numberOfVoxels, volumeDimensions, voxelSize, volumeMin, volumeCenter);

		OccupyVoxels(
			d_points,
			numberOfPoints,
			d_voxels,
			numberOfVoxels,
			volumeDimensions,
			voxelSize,
			volumeMin,
			volumeCenter,
			occupiedVoxelIndices,
			numberOfOccupiedVoxelIndices,
			occupiedPointIndices,
			numberOfOccupiedPointIndices);

		unsigned int h_numberOfOccupiedVoxelIndices = 0;
		hipMemcpy(&h_numberOfOccupiedVoxelIndices, numberOfOccupiedVoxelIndices, sizeof(unsigned int), hipMemcpyDeviceToHost);

		nvtxRangePushA("CCL");

		ConnectedComponentLabeling(d_voxels, occupiedVoxelIndices, h_numberOfOccupiedVoxelIndices, volumeDimensions);

		nvtxRangePop();
	}
	//VisualizeVoxels(
	//	d_voxels,
	//	numberOfVoxels,
	//	volumeDimensions,
	//	voxelSize,
	//	volumeMin);

	std::vector<unsigned int> result = GetLabels(
		d_points,
		numberOfPoints,
		d_voxels,
		numberOfVoxels,
		volumeDimensions,
		voxelSize,
		volumeMin,
		volumeCenter);

	{
		std::unordered_map<unsigned int, unsigned int> labelHistogram;

		for (auto& i : result)
		{
			if (0 == labelHistogram.count(i))
			{
				labelHistogram[i] = 1;
			}
			else
			{
				labelHistogram[i] += 1;
			}
		}

		unsigned int i = 0;
		for (auto& [label, count] : labelHistogram)
		{
			alog("[%4d] point label - %16d : count - %8d\n", i++, label, count);
		}
		alog("\n");
	}

	unsigned int h_numberOfOccupiedPointIndices = 0;
	hipMemcpy(&h_numberOfOccupiedPointIndices, numberOfOccupiedPointIndices, sizeof(unsigned int), hipMemcpyDeviceToHost);
	unsigned int* h_occupiedPointIndices = new unsigned int[h_numberOfOccupiedPointIndices];
	hipMemcpy(h_occupiedPointIndices, occupiedPointIndices, sizeof(unsigned int) * h_numberOfOccupiedPointIndices, hipMemcpyDeviceToHost);

	for (size_t i = 0; i < h_numberOfOccupiedPointIndices; i++)
	{
		auto index = h_occupiedPointIndices[i];
		auto p = host_points[index];

		//VD::AddSphere("In Area", { x,y, z }, 0.05f, { 255, 0, 0 });
	}

	hipFree(d_points);
	hipFree(d_voxels);
	hipFree(occupiedVoxelIndices);
	hipFree(numberOfOccupiedVoxelIndices);
	hipFree(occupiedPointIndices);
	hipFree(numberOfOccupiedPointIndices);

	delete[] h_occupiedPointIndices;

	hipDeviceSynchronize();
	nvtxRangePop();

	return result;
}